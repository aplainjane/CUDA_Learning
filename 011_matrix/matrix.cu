#include <iostream>
#include <chrono>
#include <hip/hip_runtime.h>

#include <cstdlib>  // rand

#define NX 1024
#define NY 1024
#define NXY (NX * NY)
#define BLOCK_X 16
#define BLOCK_Y 16

#define CHECK_CUDA(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        std::cout << "Error: " << hipGetErrorString(err) << std::endl; \
        exit(1); \
    } \
}

void initMatrix(float* mat, int nxy) {
    for (int i = 0; i < nxy; ++i) mat[i] = rand() / (float)RAND_MAX;
}

void transposeCPU(float* A, float* B, int nx, int ny) {
    for (int j = 0; j < ny; ++j) {
        for (int i = 0; i < nx; ++i) {
            B[i * ny + j] = A[j * nx + i];
        }
    }
}

// 上限：行读行写复制（合并满分）
__global__ void copyRow(float* A, float* B, int nx, int ny) {
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    if (ix < nx && iy < ny) {
        int idx = ix + iy * nx;  // 行索引（连续读写）
        B[idx] = A[idx];
    }
}

// 下限：列读列写复制（交叉最乱）
__global__ void copyCol(float* A, float* B, int nx, int ny) {
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    if (ix < nx && iy < ny) {
        int idx = ix * ny + iy;  // 列索引（交叉读写）
        B[idx] = A[idx];
    }
}

// Naive 转置（行读，列写）
__global__ void transposeNaive(float* A, float* B, int nx, int ny) {
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    if (ix < nx && iy < ny) {
        int idx_row = ix + iy * nx;
        int idx_col = ix * ny + iy;
        B[idx_col] = A[idx_row];
    }
}

// 展开转置（列读，行写 + 4x unroll）
__global__ void transposeUnrolled(float* A, float* B, int nx, int ny) {
    int ix = threadIdx.x + blockIdx.x * blockDim.x * 4;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    if (ix < nx && iy < ny) {
        int idx_col = ix * ny + iy;
        int idx_row = ix + iy * nx;
        B[idx_row] = A[idx_col];
        if (ix + blockDim.x < nx) B[idx_row + blockDim.x] = A[idx_col + ny * blockDim.x];
        if (ix + 2 * blockDim.x < nx) B[idx_row + 2 * blockDim.x] = A[idx_col + ny * 2 * blockDim.x];
        if (ix + 3 * blockDim.x < nx) B[idx_row + 3 * blockDim.x] = A[idx_col + ny * 3 * blockDim.x];
    }
}

int main() {
    int nBytes = NXY * sizeof(float);
    float *h_A = (float*)malloc(nBytes);
    float *h_B = (float*)malloc(nBytes);
    float *h_B_cpu = (float*)malloc(nBytes);

    initMatrix(h_A, NXY);

    float *d_A, *d_B;
    CHECK_CUDA(hipMalloc(&d_A, nBytes));
    CHECK_CUDA(hipMalloc(&d_B, nBytes));
    CHECK_CUDA(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));

    dim3 block(BLOCK_X, BLOCK_Y);
    dim3 grid((NX + BLOCK_X - 1) / BLOCK_X, (NY + BLOCK_Y - 1) / BLOCK_Y);
    dim3 grid_unroll((NX + 4 * BLOCK_X - 1) / (4 * BLOCK_X), (NY + BLOCK_Y - 1) / BLOCK_Y);

    // CPU
    auto start_cpu = std::chrono::high_resolution_clock::now();
    transposeCPU(h_A, h_B_cpu, NX, NY);
    auto end_cpu = std::chrono::high_resolution_clock::now();
    double time_cpu = std::chrono::duration<double, std::milli>(end_cpu - start_cpu).count();
    std::cout << "CPU 时间: " << time_cpu << " ms" << std::endl;

    // 上限：copyRow
    CHECK_CUDA(hipMemset(d_B, 0, nBytes));
    auto start_upper = std::chrono::high_resolution_clock::now();
    copyRow<<<grid, block>>>(d_A, d_B, NX, NY);
    CHECK_CUDA(hipDeviceSynchronize());
    auto end_upper = std::chrono::high_resolution_clock::now();
    double time_upper = std::chrono::duration<double, std::milli>(end_upper - start_upper).count();
    double bandwidth_upper = (2.0 * NXY * sizeof(float) * 1e-9) / (time_upper * 1e-3);
    std::cout << "上限 (行复制) 时间: " << time_upper << " ms, 带宽: " << bandwidth_upper << " GB/s" << std::endl;

    // 下限：copyCol
    CHECK_CUDA(hipMemset(d_B, 0, nBytes));
    auto start_lower = std::chrono::high_resolution_clock::now();
    copyCol<<<grid, block>>>(d_A, d_B, NX, NY);
    CHECK_CUDA(hipDeviceSynchronize());
    auto end_lower = std::chrono::high_resolution_clock::now();
    double time_lower = std::chrono::duration<double, std::milli>(end_lower - start_lower).count();
    double bandwidth_lower = (2.0 * NXY * sizeof(float) * 1e-9) / (time_lower * 1e-3);
    std::cout << "下限 (列复制) 时间: " << time_lower << " ms, 带宽: " << bandwidth_lower << " GB/s" << std::endl;

    // Naive
    CHECK_CUDA(hipMemset(d_B, 0, nBytes));
    auto start_naive = std::chrono::high_resolution_clock::now();
    transposeNaive<<<grid, block>>>(d_A, d_B, NX, NY);
    CHECK_CUDA(hipDeviceSynchronize());
    auto end_naive = std::chrono::high_resolution_clock::now();
    double time_naive = std::chrono::duration<double, std::milli>(end_naive - start_naive).count();
    double bandwidth_naive = (2.0 * NXY * sizeof(float) * 1e-9) / (time_naive * 1e-3);
    CHECK_CUDA(hipMemcpy(h_B, d_B, nBytes, hipMemcpyDeviceToHost));
    std::cout << "Naive 时间: " << time_naive << " ms, 带宽: " << bandwidth_naive << " GB/s" << std::endl;

    // Unrolled
    CHECK_CUDA(hipMemset(d_B, 0, nBytes));
    auto start_unroll = std::chrono::high_resolution_clock::now();
    transposeUnrolled<<<grid_unroll, block>>>(d_A, d_B, NX, NY);
    CHECK_CUDA(hipDeviceSynchronize());
    auto end_unroll = std::chrono::high_resolution_clock::now();
    double time_unroll = std::chrono::duration<double, std::milli>(end_unroll - start_unroll).count();
    double bandwidth_unroll = (2.0 * NXY * sizeof(float) * 1e-9) / (time_unroll * 1e-3);
    CHECK_CUDA(hipMemcpy(h_B, d_B, nBytes, hipMemcpyDeviceToHost));
    std::cout << "Unrolled 时间: " << time_unroll << " ms, 带宽: " << bandwidth_unroll << " GB/s (加速 " << (time_naive / time_unroll) << "x)" << std::endl;

    // 简单验证
    bool correct = true;
    for (int i = 0; i < NXY; ++i) {
        if (fabs(h_B_cpu[i] - h_B[i]) > 1e-5) { correct = false; break; }
    }
    std::cout << "验证: " << (correct ? "正确" : "错误") << std::endl;

    // 清理
    hipFree(d_A); hipFree(d_B);
    free(h_A); free(h_B); free(h_B_cpu);

    return 0;
}