#include <iostream>
#include <chrono>
#include <hip/hip_runtime.h>

#include <cstdlib>

__global__ void addKernel(float* a, float* b, float* c, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n) c[idx] = a[idx] + b[idx];
}

// 修正 sumKernel：块内共享归约 + 原子加到全局结果
__global__ void sumKernel(float* c, float* result, int n) {
    extern __shared__ float sdata[];  // 动态共享内存
    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + tid;

    // 加载到共享（边界处理）
    float sum = 0.0f;
    if (idx < n) sum = c[idx];
    sdata[tid] = sum;
    __syncthreads();

    // 块内归约（交错配对）
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) sdata[tid] += sdata[tid + s];
        __syncthreads();
    }

    // 线程0 原子加块和到全局结果
    if (tid == 0) atomicAdd(result, sdata[0]);
}

void callback(hipStream_t stream, hipError_t status, void* userData) {
    std::cout << "流1 完成！用户数据: " << *(int*)userData << std::endl;
}

#define CHECK_CUDA(call) { hipError_t err = call; if (err != hipSuccess) { std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl; exit(1); } }

int main() {
    const int N = 1024 * 1024;  // 1M 元素
    const int blockSize = 256;
    const int numBlocks = (N + blockSize - 1) / blockSize;
    size_t bytes = N * sizeof(float);

    float *h_a = new float[N], *h_b = new float[N], *h_result = new float[1];
    for (int i = 0; i < N; ++i) { h_a[i] = 1.0f; h_b[i] = 2.0f; }
    *h_result = 0.0f;

    float *d_a, *d_b, *d_c, *d_result;
    CHECK_CUDA(hipMalloc(&d_a, bytes));
    CHECK_CUDA(hipMalloc(&d_b, bytes));
    CHECK_CUDA(hipMalloc(&d_c, bytes));
    CHECK_CUDA(hipMalloc(&d_result, sizeof(float)));

    hipStream_t stream1, stream2;
    CHECK_CUDA(hipStreamCreate(&stream1));
    CHECK_CUDA(hipStreamCreate(&stream2));

    hipEvent_t start_event, end_event1, end_event2;
    CHECK_CUDA(hipEventCreate(&start_event));
    CHECK_CUDA(hipEventCreate(&end_event1));
    CHECK_CUDA(hipEventCreate(&end_event2));

    CHECK_CUDA(hipEventRecord(start_event, stream1));

    // 流1: 异步 H2D + addKernel
    CHECK_CUDA(hipMemcpyAsync(d_a, h_a, bytes, hipMemcpyHostToDevice, stream1));
    CHECK_CUDA(hipMemcpyAsync(d_b, h_b, bytes, hipMemcpyHostToDevice, stream1));
    addKernel<<<(N + 255) / 256, 256, 0, stream1>>>(d_a, d_b, d_c, N);

    CHECK_CUDA(hipEventRecord(end_event1, stream1));

    int user_data = 42;
    CHECK_CUDA(hipStreamAddCallback(stream1, callback, &user_data, 0));

    // 流2: 等待后 sumKernel（多块归约）
    CHECK_CUDA(hipStreamWaitEvent(stream2, end_event1, 0));
    sumKernel<<<numBlocks, blockSize, blockSize * sizeof(float), stream2>>>(d_c, d_result, N);

    CHECK_CUDA(hipEventRecord(end_event2, stream2));

    CHECK_CUDA(hipEventSynchronize(end_event2));

    float ms;
    CHECK_CUDA(hipEventElapsedTime(&ms, start_event, end_event1));
    std::cout << "流1 执行时间: " << ms << " ms" << std::endl;

    CHECK_CUDA(hipMemcpy(h_result, d_result, sizeof(float), hipMemcpyDeviceToHost));

    std::cout << "结果: " << *h_result << " (预期 ~3M)" << std::endl;

    delete[] h_a; delete[] h_b; delete[] h_result;
    CHECK_CUDA(hipFree(d_a)); CHECK_CUDA(hipFree(d_b)); CHECK_CUDA(hipFree(d_c)); CHECK_CUDA(hipFree(d_result));
    CHECK_CUDA(hipStreamDestroy(stream1));
    CHECK_CUDA(hipStreamDestroy(stream2));
    CHECK_CUDA(hipEventDestroy(start_event));
    CHECK_CUDA(hipEventDestroy(end_event1));
    CHECK_CUDA(hipEventDestroy(end_event2));

    return 0;
}