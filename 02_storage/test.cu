#include<hip/hip_runtime.h>
#include<iostream>
#include<cstring>
#include<cmath>

#define CHECK(call){ \
    hipError_t error = call; \
    if(error != hipSuccess){ \
        std::cout << "Error: " << hipGetErrorString(error) << std::endl; \
        exit(-1); \
    } \
} \

void initialalize(float *a, int size){
    for(int i = 0; i < size; i++){
        a[i] = static_cast<float>(rand() % 10);
    }
}

__global__ void sumArraysGPU(float *a, float *b, float *res) {
    int i = threadIdx.x;  // 获取当前线程的索引（假设块大小等于数组大小）
    res[i] = a[i] + b[i];
}

int main(){
    int dev = 0;
    CHECK(hipSetDevice(dev));

    int size = 32;
    size_t bytes = size * sizeof(float);

    float *h_a, *h_b, *h_res;
    h_a = new float[size];
    h_b = new float[size];
    h_res = new float[size];

    initialalize(h_a, size);
    initialalize(h_b, size);

    float *d_a, *d_b, *d_res;
    CHECK(hipMalloc((void **)&d_a, bytes));
    CHECK(hipMalloc((void **)&d_b, bytes));
    CHECK(hipMalloc((void **)&d_res, bytes));

    CHECK(hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice));

    sumArraysGPU<<<1, size>>>(d_a, d_b, d_res);

    CHECK(hipDeviceSynchronize());
    CHECK(hipMemcpy(h_res, d_res, bytes, hipMemcpyDeviceToHost));

    for(int i = 0; i < size; i++){
        std::cout << h_res[i] << " ";
        std::cout << h_a[i] + h_b[i] << std::endl;
    }
    std::cout << std::endl;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_res);

    delete[] h_a;
    delete[] h_b;
    delete[] h_res;

    return 0;
}